#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <time.h>
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>

#include <assert.h>

// Tested on GTX1050

#define ELEM_PER_POINT				(1 << 5)	// E
#define ELEM_PER_POINT2				(1 << 6)	// E
#define N_ELEMS						(1 << 22)	// N
#define N_POINTS					(N_ELEMS / ELEM_PER_POINT)
#define N_POINTS2					(N_ELEMS / ELEM_PER_POINT2)

#define ARRAY_2D_WIDTH				1024
#define ARRAY_2D_HEIGHT				(N_POINTS/ARRAY_2D_WIDTH)
#define ARRAY_2D_HEIGHT2			(N_POINTS2/ARRAY_2D_WIDTH)
#define BLOCK_WIDTH					128
#define BLOCK_WIDTH2				32
#define BLOCK_HEIGHT				8
#define BLOCK_HEIGHT2				32

__int64 start, freq, end;
#define CHECK_TIME_START QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start)
#define CHECK_TIME_END(a) QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f))

#define IN
#define INOUT
#define OUT


typedef struct {
	float elem[ ELEM_PER_POINT ];
} POINT_ELEMENT;

typedef struct {
	float *elem[ ELEM_PER_POINT ];
} POINTS_SOA;

typedef struct {
	float elem[ ELEM_PER_POINT2 ];
} POINT_ELEMENT2;

typedef struct {
	float *elem[ ELEM_PER_POINT2 ];
} POINTS_SOA2;

int cal_N (int n){
	int res = 0;
	while(n!=1){
		n /= 2;
		res++;
	}
	return res;
}

__constant__ float constantBuffer[ 1000 ];

__global__ void TransformAOSKernel( INOUT POINT_ELEMENT *A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;	// index

	for (int j = 2; j <= m; j++) {
		float tmp = 1.0f / (float) j;
		for (int i = 0; i < ELEM_PER_POINT; i++) {
			A[id].elem[i] += tmp*A[id].elem[i];
		}
	}
}

__global__ void TransformAOSKernel2( INOUT POINT_ELEMENT2 *A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;

	for (int j = 2; j <= m; j++) {
		float tmp = 1.0f / (float) j;
		for (int i = 0; i < ELEM_PER_POINT2; i++) {
			A[id].elem[i] += tmp*A[id].elem[i];
		}
	}

}

__global__ void TransformSOAKernel( INOUT POINTS_SOA A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;	// index

	for (int i = 0; i < ELEM_PER_POINT; i++) {
		for (int j = 2; j <= m; j++) {
			float tmp = 1.0f / (float) j;
			A.elem[i][id] += tmp*A.elem[i][id];
		}
	}
}

__global__ void TransformSOAKernel2( INOUT POINTS_SOA2 A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;

	for (int i = 0; i < ELEM_PER_POINT2; i++) {
		for (int j = 2; j <= m; j++) {
			float tmp = 1.0f / (float) j;
			A.elem[i][id] += tmp*A.elem[i][id];
		}
	}
}

__global__ void TransformAOSwithConstantMemKernel( INOUT POINT_ELEMENT *A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;	// index
	int j;

	for (int j = 2; j <= m; j++) {
		for (int i = 0; i < ELEM_PER_POINT; i++) {
			A[id].elem[i] += constantBuffer[j-1]*A[id].elem[i];
		}
	}
}

__global__ void TransformAOSwithConstantMemKernel2( INOUT POINT_ELEMENT2 *A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	int j;

	for (int j = 2; j <= m; j++) {
		for (int i = 0; i < ELEM_PER_POINT2; i++) {
			A[id].elem[i] += constantBuffer[j-1]*A[id].elem[i];
		}
	}
}

__global__ void TransformSOAwithConstantMemKernel( INOUT POINTS_SOA A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;	// index
	int i, j;

	for ( i = 0; i < ELEM_PER_POINT; ++i ) {
		for ( j = 2; j <= m; j++ ) {
			A.elem[ i ][ id ] += constantBuffer[j-1]*A.elem[ i ][ id ];
		}
	}
}

__global__ void TransformSOAwithConstantMemKernel2( INOUT POINTS_SOA2 A, IN int m )
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int id = gridDim.x*blockDim.x*row + col;
	int i, j;

	for ( i = 0; i < ELEM_PER_POINT2; ++i ) {
		for ( j = 2; j <= m; j++ ) {
			A.elem[ i ][ id ] += constantBuffer[j-1]*A.elem[ i ][ id ];
		}
	}
}

//E 16, N 22, M 128,8
void transform_points_AOS( INOUT POINT_ELEMENT *p_AOS, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	POINT_ELEMENT *d_pAOS;
	size_t size = N_POINTS * sizeof(POINT_ELEMENT);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock (BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH / dimBlock.x, ARRAY_2D_HEIGHT / dimBlock.y);

	hipEventRecord( start, 0 );	// time record start
	TransformAOSKernel <<<dimGrid, dimBlock>>> (d_pAOS, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );		// time record stop
	hipEventSynchronize( stop );

	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);	// free allocation

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("*** GPU 01 - E:%d, N:%d, B:(%d,%d) M:global, AOS : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 16, N 22, M 128,8
void transform_points_SOA( INOUT POINTS_SOA p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINTS_SOA d_pSOA;
	size_t size = N_POINTS * sizeof(float);

	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT/dimBlock.y);

	hipEventRecord( start, 0 );	// time record start
	TransformSOAKernel <<<dimGrid, dimBlock>>> (d_pSOA, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );		// time record stop
	hipEventSynchronize( stop );

	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);
	}
	for(int i=0;i<ELEM_PER_POINT;i++){
		hipFree(p_SOA.elem[i]);
	}

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("*** GPU 02 - E:%d, N:%d, B:(%d,%d) M:global, SOA : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 32, N 22, M 128,8
void transform_points_AOS3( INOUT POINT_ELEMENT2 *p_AOS, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINT_ELEMENT2 *d_pAOS;
	size_t size = N_POINTS2 * sizeof(POINT_ELEMENT2);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock (BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH / dimBlock.x, ARRAY_2D_HEIGHT2 / dimBlock.y);

	hipEventRecord( start, 0 );

	TransformAOSKernel2 <<<dimGrid, dimBlock>>> (d_pAOS, m);		// Kernel ȣ��

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("*** GPU 03 - E:%d, N:%d, B:(%d,%d) M:global, AOS : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 32, N 22, M 128,8
void transform_points_SOA3( INOUT POINTS_SOA2 p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINTS_SOA2 d_pSOA;
	size_t size = N_POINTS2 * sizeof(float);

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT2/dimBlock.y);

	hipEventRecord( start, 0 );
	TransformSOAKernel2 <<<dimGrid, dimBlock>>> (d_pSOA, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);
	}
	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipFree(p_SOA.elem[i]);
	}

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("*** GPU 04 - E:%d, N:%d, B:(%d,%d) M:global, SOA : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 32, N 22, M 32,32
void transform_points_AOS2( INOUT POINT_ELEMENT2 *p_AOS, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	

	// mycode
	POINT_ELEMENT2 *d_pAOS;
	size_t size = N_POINTS2 * sizeof(POINT_ELEMENT2);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock (BLOCK_WIDTH2, BLOCK_HEIGHT2);
	dim3 dimGrid (ARRAY_2D_WIDTH / dimBlock.x, ARRAY_2D_HEIGHT2 / dimBlock.y);



	hipEventRecord( start, 0 );
	//

//.............
	
	TransformAOSKernel2 <<<dimGrid, dimBlock>>> (d_pAOS, m);		// Kernel ȣ��

	//hipDeviceSynchronize(); //It may stall the GPU pipeline.
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	//
//.............
	//mycode
	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);

//
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("*** Array of structure: GPU Time taken = %.3fms\n", elapsedTime);
	printf("*** GPU 05 - E:%d, N:%d, B:(%d,%d) M:global, AOS : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH2, BLOCK_HEIGHT2, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 32, N 22, M 32,32
void transform_points_SOA2( INOUT POINTS_SOA2 p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINTS_SOA2 d_pSOA;
	size_t size = N_POINTS2 * sizeof(float);

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	
	dim3 dimBlock(BLOCK_WIDTH2, BLOCK_HEIGHT2);
	dim3 dimGrid (ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT2/dimBlock.y);

	hipEventRecord( start, 0 );
	TransformSOAKernel2 <<<dimGrid, dimBlock>>> (d_pSOA, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);
	}
	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipFree(p_SOA.elem[i]);
	}

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("*** Structure of array: GPU Time taken = %.3fms\n", elapsedTime);
	printf("*** GPU 06 - E:%d, N:%d, B:(%d,%d) M:global, SOA : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH2, BLOCK_HEIGHT2, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 16, N 22, M 128,8
void transform_points_AOS_with_constant( INOUT POINT_ELEMENT *p_AOS, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINT_ELEMENT *d_pAOS;
	size_t size = N_POINTS * sizeof(POINT_ELEMENT);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock (BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH / dimBlock.x, ARRAY_2D_HEIGHT / dimBlock.y);

	hipEventRecord( start, 0 );	// time record start
	TransformAOSwithConstantMemKernel <<<dimGrid, dimBlock>>> (d_pAOS, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );		// time record stop
	hipEventSynchronize( stop );

	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("*** GPU 07 - E:%d, N:%d, B:(%d,%d) M:constant, AOS : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 16, N 22, M 128,8
void transform_points_SOA_with_constant( INOUT POINTS_SOA p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINTS_SOA d_pSOA;
	size_t size = N_POINTS * sizeof(float);

	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT/dimBlock.y);

	hipEventRecord( start, 0 );	// time record start
	TransformSOAwithConstantMemKernel <<<dimGrid, dimBlock>>> (d_pSOA, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );		// time record stop
	hipEventSynchronize( stop );

	for(int i=0;i<ELEM_PER_POINT;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);
	}
	for(int i=0;i<ELEM_PER_POINT;i++){
		hipFree(p_SOA.elem[i]);
	}

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	printf("*** GPU 08 - E:%d, N:%d, B:(%d,%d) M:constant, SOA : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 32, N 22, M 128,8
void transform_points_AOS_with_constant3( INOUT POINT_ELEMENT2 *p_AOS, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	

	// mycode
	POINT_ELEMENT2 *d_pAOS;
	size_t size = N_POINTS2 * sizeof(POINT_ELEMENT2);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock (BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH / dimBlock.x, ARRAY_2D_HEIGHT2 / dimBlock.y);



	hipEventRecord( start, 0 );
	//

//.............
	
	TransformAOSwithConstantMemKernel2 <<<dimGrid, dimBlock>>> (d_pAOS, m);		// Kernel ȣ��

	//hipDeviceSynchronize(); //It may stall the GPU pipeline.
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	//
//.............
	//mycode
	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);

//
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("*** Array of structure: GPU Time taken = %.3fms\n", elapsedTime);
	printf("*** GPU 09 - E:%d, N:%d, B:(%d,%d) M:constant, AOS : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);


}

//E 32, N 22, M 128,8
void transform_points_SOA_with_constant3( INOUT POINTS_SOA2 p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINTS_SOA2 d_pSOA;
	size_t size = N_POINTS2 * sizeof(float);

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 dimGrid (ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT2/dimBlock.y);

	hipEventRecord( start, 0 );
	TransformSOAwithConstantMemKernel2 <<<dimGrid, dimBlock>>> (d_pSOA, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);
	}
	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipFree(p_SOA.elem[i]);
	}

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("*** Structure of array: GPU Time taken = %.3fms\n", elapsedTime);
	printf("*** GPU 10 - E:%d, N:%d, B:(%d,%d) M:constant, SOA : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH, BLOCK_HEIGHT, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//E 32, N 22, M 32,32
void transform_points_AOS_with_constant2( INOUT POINT_ELEMENT2 *p_AOS, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	

	// mycode
	POINT_ELEMENT2 *d_pAOS;
	size_t size = N_POINTS * sizeof(POINT_ELEMENT2);
	hipMalloc(&d_pAOS, size);
	hipMemcpy(d_pAOS, p_AOS, size, hipMemcpyHostToDevice);
	dim3 dimBlock (BLOCK_WIDTH2, BLOCK_HEIGHT2);
	dim3 dimGrid (ARRAY_2D_WIDTH / dimBlock.x, ARRAY_2D_HEIGHT2 / dimBlock.y);



	hipEventRecord( start, 0 );
	//

//.............
	
	TransformAOSwithConstantMemKernel2 <<<dimGrid, dimBlock>>> (d_pAOS, m);		// Kernel ȣ��

	//hipDeviceSynchronize(); //It may stall the GPU pipeline.
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	//
//.............
	//mycode
	hipMemcpy(p_AOS, d_pAOS, size, hipMemcpyDeviceToHost);
	hipFree(d_pAOS);

//
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("*** Array of structure: GPU Time taken = %.3fms\n", elapsedTime);
	printf("*** GPU 11 - E:%d, N:%d, B:(%d,%d) M:constant, AOS : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH2, BLOCK_HEIGHT2, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);


}

//E 32, N 22, M 32,32
void transform_points_SOA_with_constant2( INOUT POINTS_SOA2 p_SOA, IN int n_points, IN int m )
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	POINTS_SOA2 d_pSOA;
	size_t size = N_POINTS * sizeof(float);

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMalloc(&d_pSOA.elem[i], size);
		hipMemcpy(d_pSOA.elem[i], p_SOA.elem[i], size, hipMemcpyHostToDevice);
	}
	
	dim3 dimBlock(BLOCK_WIDTH2, BLOCK_HEIGHT2);
	dim3 dimGrid (ARRAY_2D_WIDTH/dimBlock.x, ARRAY_2D_HEIGHT2/dimBlock.y);

	hipEventRecord( start, 0 );
	TransformSOAwithConstantMemKernel2 <<<dimGrid, dimBlock>>> (d_pSOA, m);		// Kernel ȣ��
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipMemcpy(p_SOA.elem[i], d_pSOA.elem[i], size, hipMemcpyDeviceToHost);
	}
	for(int i=0;i<ELEM_PER_POINT2;i++){
		hipFree(p_SOA.elem[i]);
	}

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
//	printf("*** Structure of array: GPU Time taken = %.3fms\n", elapsedTime);
	printf("*** GPU 12 - E:%d, N:%d, B:(%d,%d) M:constant, SOA : GPU Time taken = %.3fms\n",
		ELEM_PER_POINT2, cal_N(N_ELEMS), BLOCK_WIDTH2, BLOCK_HEIGHT2, elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

void generate_point_data( OUT POINT_ELEMENT *p_AOS, OUT POINTS_SOA *p_SOA, IN int n )
{
	int i;

	srand( ( unsigned int )time( NULL ) );

	for( i = 0; i < n; i++ )
	{
		for( int j = 0; j < ELEM_PER_POINT; ++j )
		{
			p_AOS[ i ].elem[ j ] = p_SOA->elem[ j ][ i ] = 100.0f*( ( float )rand() ) / RAND_MAX;
		}
	}
}

void generate_point_data2( OUT POINT_ELEMENT2 *p_AOS, OUT POINTS_SOA2 *p_SOA, IN int n )
{
	int i;

	srand( ( unsigned int )time( NULL ) );

	for( i = 0; i < n; i++ )
	{
		for( int j = 0; j < ELEM_PER_POINT2; ++j )
		{
			p_AOS[ i ].elem[ j ] = p_SOA->elem[ j ][ i ] = 100.0f*( ( float )rand() ) / RAND_MAX;
		}
	}
}

void generate_constant_data( IN int m )
{
	float *p_constant = new float[ m ];

	p_constant[ 0 ] = 0; // not used
	for( int i = 2; i <= m; ++i )
	{
		p_constant[ i-1 ] = 1.0f / ( float )i;
	}

	hipMemcpyToSymbol(HIP_SYMBOL( constantBuffer), p_constant, sizeof( float )* m );

	delete[] p_constant;
}

int main(void){

	float compute_time;
	int n_points, cutoff;
	POINT_ELEMENT *Points_AOS;
	POINTS_SOA Points_SOA;
	POINT_ELEMENT2 *Points_AOS2;
	POINTS_SOA2 Points_SOA2;

	n_points = N_POINTS;
	cutoff = 1000;
	Points_AOS = new POINT_ELEMENT[N_POINTS];
	Points_AOS2 = new POINT_ELEMENT2[N_POINTS];
	for( int i = 0; i < ELEM_PER_POINT; ++i )
	{
		Points_SOA.elem[ i ] = new float[ N_POINTS ];
	}
	for( int i = 0; i < ELEM_PER_POINT2; ++i )
	{
		Points_SOA2.elem[ i ] = new float[ N_POINTS ];
	}
	generate_point_data(Points_AOS, &Points_SOA, n_points);
	generate_point_data2(Points_AOS2, &Points_SOA2, n_points);
	generate_constant_data(cutoff);

	transform_points_AOS(Points_AOS, n_points, cutoff);	// E 16, N 22, M 128,8
	transform_points_SOA(Points_SOA, n_points, cutoff);	// E 16, N 22, M 128,8
	printf("\n");

	transform_points_AOS3(Points_AOS2, n_points, cutoff);	// E 32, N 22, M 128,8
	transform_points_SOA3(Points_SOA2, n_points, cutoff);	// E 32, N 22, M 128,8
	printf("\n");

	transform_points_AOS2(Points_AOS2, n_points, cutoff);	// E 32, N 22, M 32,32
	transform_points_SOA2(Points_SOA2, n_points, cutoff);	// E 32, N 22, M 32,32
	printf("\n");

	transform_points_AOS_with_constant(Points_AOS, n_points, cutoff);	// E 16, N 22, M 128,8
	transform_points_SOA_with_constant(Points_SOA, n_points, cutoff);	// E 16, N 22, M 128,8
	printf( "\n" );

	transform_points_AOS_with_constant3(Points_AOS2, n_points, cutoff);	// E 32, N 22, M 128,8
	transform_points_SOA_with_constant3(Points_SOA2, n_points, cutoff);	// E 32, N 22, M 128,8
	printf("\n");

	transform_points_AOS_with_constant2(Points_AOS2, n_points, cutoff);	// E 32, N 22, M 32,32
	transform_points_SOA_with_constant2(Points_SOA2, n_points, cutoff);	// E 32, N 22, M 32,32
	printf( "\n" );

	return 0;
}